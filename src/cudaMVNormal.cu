#include "hip/hip_runtime.h"
#include <assert.h>

#include "cudaMVNormal.hu"

/*
 * Computes \sum_{i}^{N} x_i y_i for x, y \in \mathbb{R}^{N}.
 */
__device__ float devVecDot(const size_t N, const float* x, const float* y) {
	assert(N > 0);
	assert(x != NULL);
	assert(y != NULL);
	// x == y allowed

	float sum = 0;
	for(size_t i = 0; i < N; ++i) {
		sum += x[i] * y[i];
	}
	return sum;
}

/*
 * Computes z_{i} \gets x_{i} - y_{i} for x, y \in \mathbb{R}^N.
 */
__device__ void devVecMinus(const size_t N, float* z, const float* x, const float* y) {
	assert(N > 0);
	assert(x != NULL);
	assert(y != NULL);
	// x == y allowed

	for(size_t i = 0; i < N; ++i) {
		z[i] = x[i] - y[i];
	}
}

/*
 * Solves the lower triangular system L^T x = b for x, b \in \mathbb{R}^{N}, 
 * L \in \mathbb{R}^{N \times N} and L_{i, j} = 0 for j > i.
 */
__device__ void devSolveLowerTri(const size_t N, const float* L, float* x, const float* b) {
	assert(N > 0);
	assert(L != NULL);
	assert(x != NULL);
	assert(b != NULL);
	// x == b allowed

	for(size_t i = 0; i < N; ++i) {
		float sum = 0.0;
		if(i > 0) {
			for(size_t j = 0; j <= i - 1; ++j) {
				sum += L[i * N + j] * x[j];
			}
		}

		x[i] = (b[i] - sum) / L[i * N + i];
	}
}

/*
 * Solves the upper triangular system L^T x = b for x, b \in \mathbb{R}^{N}, 
 * L \in \mathbb{R}^{N \times N} and L_{i, j} = 0 for j > i.
 */
__device__ void devSolveLowerTriT(const size_t N, const float* L, float* x, const float* b) {
	assert(N > 0);
	assert(L != NULL);
	assert(x != NULL);
	assert(b != NULL);
	// x == b allowed

	// treat L as an upper triangular matrix U
	for(size_t i = 0; i < N; i++) {
		size_t ip = N - 1 - i;
		float sum = 0;
		for(size_t j = ip + 1; j < N; ++j) {
			sum += L[j * N + ip] * x[j];
		}

		x[ip] = (b[ip] - sum) / L[ip * N + ip];
	}
}


/*
 *
 */
__device__ float devLogMVNormNormalizer(
	const size_t pointDim,
	const float* sigmaL
) {
	float det = 1.0;
	for(size_t i = 0; i < pointDim; ++i) {
		det *= sigmaL[i * pointDim + i];
	}
	det *= det;

	return -0.5 * logf( 2.0 * M_PI ) * pointDim - 0.5 * logf(det);
}

/*
 * Computes logf( p(x | mu, Sigma ) ) for multivariate normal distribution with 
 * parameters mu (mean), and Sigma (covariance).
 */
__device__ float devLogMVNormDist(
	const size_t pointDim,
	const float* x, const float* mu, const float* sigmaL,
	float* u, float* v
) {
	devVecMinus(pointDim, v, x, mu); // v <- x - mu
	devSolveLowerTri(pointDim, sigmaL, u, v); // u <- u s.t. L u = (x - mu)
	devSolveLowerTriT(pointDim, sigmaL, u, u); // u <- v s.t. L^T v = u
	return devLogMVNormNormalizer(pointDim, sigmaL) - 0.5 * devVecDot(pointDim, u, v);
}

__global__ void kernLogMVNormDist(
	const size_t numPoints, const size_t pointDim, 
	const float* X, float* mu, float* sigmaL,
	float* logProb
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	float u[1024];
	float v[1024];

	logProb[i] = devLogMVNormDist(
		pointDim, 
		& X[i * pointDim], mu, sigmaL,
		u, v
	);
}


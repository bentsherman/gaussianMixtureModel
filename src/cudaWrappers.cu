#include "hip/hip_runtime.h"
#include <assert.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdexcept>

// Intentionally not including header since it is meant for gcc consumption.
// #include "cudaWrappers.h"

#include "cudaCommon.hu"
#include "cudaFolds.hu"
#include "cudaGmm.hu"
#include "cudaMVNormal.hu"
#include "gmm.h"

void gpuSum(size_t numPoints, const size_t pointDim, float* host_a, float* host_sum) {
	assert(numPoints > 0);
	assert(pointDim > 0);
	assert(host_a != NULL);
	assert(host_sum != NULL);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	// cudaArraySum is meant for powers of two
	size_t M = largestPowTwoLessThanEq(numPoints);

	float cpuSum[pointDim];
	memset(cpuSum, 0, pointDim * sizeof(float));
	for(size_t i = M; i < numPoints; ++i) {
		for(size_t j = 0; j < pointDim; ++j) {
			cpuSum[j] += host_a[i * pointDim + j];
		}
	}

	numPoints = M;

	float *device_a = sendToGpu(numPoints * pointDim, host_a);

	// cudaArraySum is synchronous
	cudaArraySum(
		&deviceProp, numPoints, pointDim, device_a
		);

	check(hipMemcpy(host_sum, device_a, pointDim * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(device_a);

	for(size_t i = 0; i < pointDim; ++i) {
		host_sum[i] += cpuSum[i];
	}
}

float gpuMax(size_t N, float* host_a) {
	assert(host_a != NULL);
	assert(N > 0);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	float *device_a = sendToGpu(N, host_a);

	cudaArrayMax(
		&deviceProp, N, device_a
		);

	float gpuMax = 0;
	check(hipMemcpy(&gpuMax, device_a, sizeof(float), hipMemcpyDeviceToHost));

	hipFree(device_a);

	return gpuMax;
}

void gpuLogMVNormDist(
	const size_t numPoints, const size_t pointDim,
	const float* X, const float* mu, const float* sigmaL,
	float* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	float* device_X = sendToGpu(numPoints * pointDim, X);
	float* device_mu = sendToGpu(pointDim, mu);
	float* device_sigmaL = sendToGpu(pointDim * pointDim, sigmaL);
	float* device_logP = mallocOnGpu(numPoints);

	dim3 grid, block;
	calcDim(numPoints, &deviceProp, &block, &grid);
	kernLogMVNormDist<<<grid, block>>>(
		numPoints, pointDim,
		device_X, device_mu, device_sigmaL,
		device_logP
		);

	check(hipMemcpy(logP, device_logP, numPoints * sizeof(float), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	hipFree(device_X);
	hipFree(device_mu);
	hipFree(device_sigmaL);
	hipFree(device_logP);
}

float gpuGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const float* logpi, float* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	float* device_logpi = sendToGpu(numComponents, logpi);

	// Sending all data because logP is an array organized by:
	// [ <- numPoints -> ]_0 [ <- numPoints -> ]_... [ <- numPoints -> ]_{k-1}
	// So even though we are only using M of those points on the GPU,
	// we need all numPoints to ensure indexing by numPoints * k + i works
	// correctly to access prob(x_i|mu_k,Sigma_k).
	float* device_logP = sendToGpu(numComponents * numPoints, logP);

	float logL = cudaGmmLogLikelihoodAndGammaNK(
		& deviceProp,
		numPoints, numComponents,
		logpi, logP,
		device_logpi, device_logP
	);

	hipFree(device_logpi);
	hipFree(device_logP);

	return logL;
}

void gpuCalcLogGammaNK(
	const size_t numPoints, const size_t numComponents,
	const float* logpi, float* loggamma
) {
	gpuGmmLogLikelihood(
		numPoints, numComponents,
		logpi, loggamma
	);
}

void gpuCalcLogGammaK(
	const size_t numPoints, const size_t numComponents,
	const float* loggamma, float* logGamma
) {
	// Gamma[k] = max + log sum expf(loggamma - max)

	float* working = (float*)malloc(numPoints * sizeof(float));
	for(size_t k = 0; k < numComponents; ++k) {
		// TODO: refactor to have a generic z = a + log sum expf(x - a)
		memcpy(working, & loggamma[k * numPoints], numPoints * sizeof(float));
		float maxValue = gpuMax(numPoints, working);

		memcpy(working, & loggamma[k * numPoints], numPoints * sizeof(float));
		for(size_t i = 0; i < numPoints; ++i) {
			working[i] = expf(working[i] - maxValue);
		}

		float sum = 0;
		gpuSum(numPoints, 1, working, & sum);
 		logGamma[k] = maxValue + logf(sum );
	}
	free(working);
}


void gpuGmmFit(
	const float* X,
	const size_t numPoints,
	const size_t pointDim,
	const size_t numComponents,
	float* pi,
	float* Mu,
	float* Sigma,
	float* SigmaL,
	float* normalizers,
	const size_t maxIterations,
	GMM* gmm
) {
	assert(X != NULL);
	assert(numPoints > 0);
	assert(pointDim > 0 && pointDim <= 1024);
	assert(numComponents > 0 && numComponents <= 1024);

	assert(pi != NULL);
	assert(Mu != NULL);
	assert(Sigma != NULL);
	assert(SigmaL != NULL);
	assert(normalizers != NULL);

	assert(maxIterations >= 1);

	int deviceId;

	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	// printf("name: %s\n", deviceProp.name);
	// printf("multiProcessorCount: %d\n", deviceProp.multiProcessorCount);
	// printf("concurrentKernels: %d\n", deviceProp.concurrentKernels);

	float* device_X = pinHostAndSendDevice(numPoints * pointDim, (float*) X);

	for(size_t i = 0; i < numComponents; ++i) {
		assert(pi[i] > 0);
		pi[i] = logf(pi[i]);
	}

	float* device_logpi = pinHostAndSendDevice(numComponents, pi);
	float* device_Mu = pinHostAndSendDevice(numComponents * pointDim, Mu);
	float* device_Sigma = pinHostAndSendDevice(numComponents * pointDim * pointDim, Sigma);

	float* device_SigmaL = pinHostAndSendDevice(numComponents * pointDim * pointDim, SigmaL);
	float* device_normalizers = pinHostAndSendDevice(numComponents, normalizers);

	int error = 0;
	int* device_error = (int*) pinHostAndSendDevice(1, (float*) &error);

	float* device_loggamma = mallocOnGpu(numPoints * numComponents);
	float* device_logGamma = mallocOnGpu(numPoints * numComponents);

	float previousLogL = -INFINITY;

	float* pinnedCurrentLogL;
	hipHostMalloc(&pinnedCurrentLogL, sizeof(float));
	*pinnedCurrentLogL = -INFINITY;

	// logPx, mu, sigma reductions
	// This means for mu and sigma can only do one component at a time otherwise
	// the memory foot print will limit how much data we can actually work with.
	float* device_working = mallocOnGpu(numComponents * numPoints * pointDim * pointDim);

	dim3 grid, block;
	calcDim(numPoints, &deviceProp, &block, &grid);

	size_t iteration = 0;
	const float tolerance = 1e-8;

	hipStream_t streams[numComponents];
	for(size_t k = 0; k < numComponents; ++k) {
		hipStreamCreate(&streams[k]);
	}

	hipEvent_t kernelEvent[numComponents + 1];
	for(size_t k = 0; k <= numComponents; ++k) {
		hipEventCreateWithFlags(& kernelEvent[k], hipEventDisableTiming);
	}

	try {
		do {
			// --------------------------------------------------------------------------
			// E-Step
			// --------------------------------------------------------------------------

			// loggamma[k * numPoints + i] = p(x_i | mu_k, Sigma_k )
			for(size_t k = 0; k < numComponents; ++k) {
				// Fill in numPoint many probabilities
				kernLogMVNormDist<<<grid, block, 0, streams[k]>>>(
					numPoints, pointDim,
					device_X,
					& device_Mu[k * pointDim],
					& device_SigmaL[k * pointDim * pointDim],
					& device_loggamma[k * numPoints]
				);

				hipEventRecord(kernelEvent[k], streams[k]);
			}

			for(size_t k = 0; k < numComponents; ++k) {
				// streams[numComponents - 1] needs to wait for everyone else to finish
				hipStreamWaitEvent(streams[numComponents-1], kernelEvent[k], 0);
			}

			// loggamma[k * numPoints + i] = p(x_i | mu_k, Sigma_k) / p(x_i)
			// working[i] = p(x_i)
			kernCalcLogLikelihoodAndGammaNK<<<grid, block, 0, streams[numComponents - 1]>>>(
				numPoints, numComponents,
				device_logpi, device_working, device_loggamma
			);

			// working[0] = sum_{i} p(x_i)
			cudaArraySum(&deviceProp, numPoints, 1, device_working, streams[numComponents - 1]);

			previousLogL = *pinnedCurrentLogL;
			check(hipMemcpyAsync(
				pinnedCurrentLogL, device_working,
				sizeof(float),
				hipMemcpyDeviceToHost,
				streams[numComponents - 1]
			));

			for(size_t k = 0; k < numComponents; ++k) {
				// synchronize everybody with the host
				hipStreamSynchronize(streams[k]);
			}

			if(fabsf(*pinnedCurrentLogL - previousLogL) < tolerance || *pinnedCurrentLogL < previousLogL) {
				break;
			}

			// --------------------------------------------------------------------------
			// M-Step
			// --------------------------------------------------------------------------

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				cudaLogSumExp(
					& deviceProp, grid, block,
					numPoints,
					& device_loggamma[k * numPoints], & device_logGamma[k * numPoints],
					device_workingK,
					streams[k]
				);
			}

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				// working[i * pointDim + j] = gamma_ik / Gamma K * x_j
				kernCalcMu<<<grid, block, 0, streams[k]>>>(
					numPoints, pointDim,
					device_X,
					& device_loggamma[k * numPoints],
					& device_logGamma[k * numPoints],
					device_workingK
				);
			}

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				// working[0 + j] = sum gamma_ik / Gamma K * x_j
				cudaArraySum(
					&deviceProp, numPoints, pointDim,
					device_workingK,
					streams[k]
				);
			}

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				check(hipMemcpyAsync(
					& device_Mu[k * pointDim],
					device_workingK,
					pointDim * sizeof(float),
					hipMemcpyDeviceToDevice,
					streams[k]
				));
			}

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				check(hipMemcpyAsync(
					& device_Sigma[k * pointDim * pointDim],
					device_workingK,
					pointDim * pointDim * sizeof(float),
					hipMemcpyDeviceToDevice,
					streams[k]
				));
			}

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				kernCalcSigma<<<grid, block, 0, streams[k]>>>(
					numPoints, pointDim,
					device_X,
					& device_Mu[k * pointDim],
					& device_loggamma[k * numPoints],
					& device_logGamma[k * numPoints],
					device_workingK
				);
			}

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				// working[0 + j] = sum gamma_ik / Gamma K * [...]_j
				cudaArraySum(
					&deviceProp, numPoints, pointDim * pointDim,
					device_workingK,
					streams[k]
				);
			}

			for(size_t k = 0; k < numComponents; ++k) {
				float* device_workingK = & device_working[k * numPoints * pointDim * pointDim];
				check(hipMemcpyAsync(
					& device_Sigma[k * pointDim * pointDim],
					device_workingK,
					pointDim * pointDim * sizeof(float),
					hipMemcpyDeviceToDevice,
					streams[k]
				));

				hipEventRecord(kernelEvent[k], streams[k]);
			}

			for(size_t k = 0; k < numComponents; ++k) {
				// streams[numComponents - 1] needs to wait for everyone else to finish
				hipStreamWaitEvent(streams[numComponents-1], kernelEvent[k], 0);
			}

			// pi_k^(t+1) = pi_k Gamma_k / sum_{i}^{K} pi_i * Gamma_i
			// Use thread sync to compute denom to avoid data race
			kernUpdatePi<<<1, numComponents, 0, streams[numComponents - 1]>>>(
				numPoints, numComponents,
				device_logpi, device_logGamma
			);

			// recompute sigmaL and normalizer
			kernPrepareCovariances<<<1, numComponents, 0, streams[numComponents - 1]>>>(
				numComponents, pointDim,
				device_Sigma, device_SigmaL,
				device_normalizers,
				device_error
			);

			hipEventRecord(kernelEvent[numComponents], streams[numComponents - 1]);

			for(size_t k = 0; k < numComponents; ++k) {
				// Everyone needs to wait for the work on streams[numComponents - 1] to finish.
				hipStreamWaitEvent(streams[k], kernelEvent[numComponents], 0);
			}

			// check error to see if inverse failed
			check(hipMemcpy(&error, device_error, sizeof(int), hipMemcpyDeviceToHost));

			if ( error ) {
				throw std::runtime_error("Failed to compute inverse");
			}

		} while(++iteration < maxIterations);

		// copy loggamma to host to compute output labels
		float* loggamma = (float *)malloc(numPoints * numComponents * sizeof(float));

		check(hipMemcpy(
			loggamma,
			device_loggamma,
			numPoints * numComponents * sizeof(float),
			hipMemcpyDeviceToHost
		));

		gmm->failed = false;
		gmm->y_pred = calcLabels(loggamma, numPoints, numComponents);
		gmm->logL = *pinnedCurrentLogL;
	}
	catch ( std::runtime_error& e ) {
		fprintf(stderr, "warning: model failed\n");
		gmm->failed = true;
	}

	for(size_t k = 0; k <= numComponents; ++k) {
		hipEventDestroy(kernelEvent[k]);
	}

	for(size_t k = 0; k < numComponents; ++k) {
		hipStreamDestroy(streams[k]);
	}

	hipHostFree(pinnedCurrentLogL);
	hipFree(device_working);
	hipFree(device_logGamma);
	hipFree(device_loggamma);

	unpinHost(device_error, &error);
	recvDeviceUnpinHost(device_normalizers, normalizers, numComponents);
	recvDeviceUnpinHost(device_SigmaL, SigmaL, numComponents * pointDim * pointDim);
	recvDeviceUnpinHost(device_Sigma, Sigma, numComponents * pointDim * pointDim);
	recvDeviceUnpinHost(device_Mu, Mu, numComponents * pointDim);
	recvDeviceUnpinHost(device_logpi, pi, numComponents);

	for(size_t i = 0; i < numComponents; ++i) {
		pi[i] = expf(pi[i]);
	}

	unpinHost(device_X, (float*) X);
}

#include "hip/hip_runtime.h"
#include <assert.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"
#include "cudaFolds.hu"
#include "cudaGmm.hu"

__global__ void kernCalcLogLikelihoodAndGammaNK(
	const size_t numPoints, const size_t numComponents,
	const float* logpi, float* logPx, float* loggamma
) {
	// loggamma[k * numPoints + i] =
	// On Entry: log p(x_i | mu_k, Sigma_k)
	// On exit: [log pi_k] + [log p(x_i | mu_k, sigma_k)] - [log p(x_i)]

	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	float maxArg = -INFINITY;
	for(size_t k = 0; k < numComponents; ++k) {
		const float logProbK = logpi[k] + loggamma[k * numPoints + i];
		if(logProbK > maxArg) {
			maxArg = logProbK;
		}
	}

	float sum = 0.0;
	for (size_t k = 0; k < numComponents; ++k) {
		const float logProbK = logpi[k] + loggamma[k * numPoints + i];
		sum += expf(logProbK - maxArg);
	}

	assert(sum >= 0);
	const float logpx = maxArg + logf(sum);

	for(size_t k = 0; k < numComponents; ++k) {
		loggamma[k * numPoints + i] += -logpx;
	}

	logPx[i] = logpx;
}

__host__ float cudaGmmLogLikelihoodAndGammaNK(
	hipDeviceProp_t* deviceProp,
	const size_t numPoints, const size_t numComponents,
	const float* logpi, float* logP,
	const float* device_logpi, float* device_logP
) {
	// logpi: 1 x numComponents
	// logP: numComponents x numPoints

	dim3 grid, block;
	calcDim(numPoints, deviceProp, &block, &grid);

	float logL = 0;
	float* device_logPx = mallocOnGpu(numPoints);

	kernCalcLogLikelihoodAndGammaNK<<<grid, block>>>(
		numPoints, numComponents,
		device_logpi, device_logPx, device_logP
	);

	cudaArraySum(
		deviceProp,
		numPoints, 1,
		device_logPx
	);

	check(hipMemcpy(&logL, device_logPx, sizeof(float), hipMemcpyDeviceToHost));

	hipFree(device_logPx);

	// Copy back the full numPoints * numComponents values
	check(hipMemcpy(logP, device_logP,
		numPoints * numComponents * sizeof(float), hipMemcpyDeviceToHost));

	return logL;
}

__global__ void kernExp(float* A, float* bias) {
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	A[i] = expf(A[i] - *bias);
}

__global__ void kernBiasAndLog(float* sumexp, float* bias) {
	*sumexp = *bias + logf(*sumexp);
}

__host__ void cudaLogSumExp(
	hipDeviceProp_t* deviceProp, dim3 grid, dim3 block,
	const size_t numPoints,
	float* device_src, float* device_dest,
	float* device_working,
	hipStream_t stream
) {
	// dest <- src
	check(hipMemcpyAsync(
		device_dest, device_src,
		numPoints * sizeof(float),
		hipMemcpyDeviceToDevice,
		stream
	));

	// working <- src
	check(hipMemcpyAsync(
		device_working, device_src,
		numPoints * sizeof(float),
		hipMemcpyDeviceToDevice,
		stream
	));

	// working <- max { src }
	cudaArrayMax(deviceProp, numPoints, device_working, stream);

	// dest <- expf(src - max)
	kernExp<<<grid, block, 0, stream>>>(
		device_dest,
		device_working
	);

	// dest <- sum expf(src - max)
	cudaArraySum(deviceProp, numPoints, 1, device_dest, stream);

	// dest <- max + log sum expf(src - max)
	kernBiasAndLog<<<1, 1, 0, stream>>>(
		device_dest, device_working
	);
}

__global__ void kernCalcMu(
	const size_t numPoints, const size_t pointDim,
	const float* X, const float* loggamma, const float* GammaK,
	float* dest
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	const float a = expf(loggamma[i]) / expf(*GammaK);
	const float* x = & X[i * pointDim];
	float* y = & dest[i * pointDim];

	for(size_t i = 0; i < pointDim; ++i) {
		y[i] = a * x[i];
	}
}

__global__ void kernCalcSigma(
	const size_t numPoints, const size_t pointDim,
	const float* X, const float* mu, const float* loggamma, const float* GammaK,
	float* dest
) {
	assert(pointDim < 1024);

	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	// gamma_{n,k} / Gamma_{k} (x - mu) (x - mu)^T

	const float a = expf(loggamma[i]) / expf(*GammaK);
	const float* x = & X[i * pointDim];
	float* y = & dest[i * pointDim * pointDim];

	float u[1024];
	for(size_t i = 0; i < pointDim; ++i) {
		u[i] = x[i] - mu[i];
	}

	for(size_t i = 0; i < pointDim; ++i) {
		float* yp = &y[i * pointDim];
		for(size_t j = 0; j < pointDim; ++j) {
			yp[j] = a * u[i] * u[j];
		}
	}
}

__global__ void kernUpdatePi(
	const size_t numPoints, const size_t numComponents,
	float* logpi, float* Gamma
) {
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int comp = b * blockDim.x + threadIdx.x;
	if(comp > numComponents) {
		return;
	}

	__shared__ float A[1024];
	A[comp] = logpi[comp] + logf(Gamma[comp * numPoints]);
	__syncthreads();

	float sum = 0;
	for(size_t k = 0; k < numComponents; ++k) {
		sum += expf(A[k]);
	}

	logpi[comp] = A[comp] - logf(sum);
}

__global__ void kernPrepareCovariances(
	const size_t numComponents, const size_t pointDim,
	float* Sigma, float* SigmaL,
	float* normalizers,
	int *error
) {
	// Parallel in the number of components

	// Sigma: numComponents x pointDim * pointDim
	// SigmaL: numComponents x pointDim * pointDim
	// normalizers: 1 x numComponents

	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int comp = b * blockDim.x + threadIdx.x;
	if(comp > numComponents) {
		return;
	}

	// L is the resulting lower diagonal portion of A = LL^T
	const size_t ALen = pointDim * pointDim;
	float* A = & Sigma[comp * ALen];
	float* L = & SigmaL[comp * ALen];
	for(size_t i = 0; i < ALen; ++i) {
		L[i] = 0;
	}

	for (size_t k = 0; k < pointDim; ++k) {
		float sum = 0;
		for (int s = 0; s < k; ++s) {
			const float l = L[k * pointDim + s];
			const float ll = l * l;
			sum += ll;
		}

		assert(sum >= 0);

		sum = A[k * pointDim + k] - sum;
		if (sum <= FLT_EPSILON) {
			*error = 1;
			return;
		}

		L[k * pointDim + k] = sqrtf(sum);
		for (int i = k + 1; i < pointDim; ++i) {
			float subsum = 0;
			for (int s = 0; s < k; ++s)
				subsum += L[i * pointDim + s] * L[k * pointDim + s];

			L[i * pointDim + k] = (A[i * pointDim + k] - subsum) / L[k * pointDim + k];
		}
	}

	float logDet = 1.0;
	for (size_t i = 0; i < pointDim; ++i) {
		float diag = L[i * pointDim + i];
		assert(diag > 0);
		logDet += logf(diag);
	}

	logDet *= 2.0;

	normalizers[comp] = - 0.5 * (pointDim * logf(2.0 * M_PI) + logDet);
}

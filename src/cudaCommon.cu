#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"

__host__ void assertPowerOfTwo(size_t N) {
	int bit = 0;
	while(N > 0 && bit <= 1) {
		bit += N & 1;
		N >>= 1;
	}

	assert(bit <= 1);
}

__host__ size_t largestPowTwoLessThanEq(size_t N) {
	// Assigns the largest value (M = 2^n) < N to N and returns the residual.
	if(N == 0) {
		return 0;
	} // PC: N > 0

	size_t M = 1;
	while(M < N) {
		M *= 2;
	} // PC: M >= N

	if(M == N) {
		return M;
	} // PC: M > N
	
	return M / 2;
}

__host__ void calcDim(int N, hipDeviceProp_t* devProp, dim3* block, dim3* grid) {
	assert(devProp != NULL);
	assert(block != NULL);
	assert(grid != NULL);

	// make a 2D grid of 1D blocks
	const int numThreadRows = 1;
	const int numThreadCols = devProp->maxThreadsPerBlock;
	block->x = min(numThreadCols, N);
	block->y = numThreadRows;

	const int numThreadsPerBlock = numThreadRows * numThreadCols;
	const int residualThreads = N % numThreadsPerBlock;
	int numBlocksPerGrid = (N - residualThreads) / numThreadsPerBlock;
	if(residualThreads > 0) {
		++numBlocksPerGrid;
	}

	const int numBlockCols = min( numBlocksPerGrid, devProp->maxGridSize[0] );
	const int residualBlocks = numBlocksPerGrid % numBlockCols;
	int numBlockRows = (numBlocksPerGrid - residualBlocks) / numBlockCols;
	if(residualBlocks > 0) {
		++numBlockRows;
	}

	grid->x = numBlockCols;
	grid->y = numBlockRows;

	assert(block->x * block->y * block->z > 0);
	assert(block->x * block->y * block->z <= devProp->maxThreadsPerBlock);

	assert(grid->x <= devProp->maxGridSize[0]);
	assert(grid->y <= devProp->maxGridSize[1]);
	assert(grid->z <= devProp->maxGridSize[2]);
}

__host__ void dimToConsole(dim3* block, dim3* grid) {
	assert(block != NULL);
	assert(grid != NULL);

	printf("block: (%d, %d, %d)\n", block->x, block->y, block->z);
	printf("grid: (%d, %d, %d)\n", grid->x, grid->y, grid->z);
}

__host__ float* mallocOnGpu(const size_t N) {
	float* device_A;
	float ABytes = N * sizeof(float);
	check(hipMalloc(&device_A, ABytes));
	return device_A;
}

__host__ float* sendToGpu(const size_t N, const float* host) {
	float* device;
	const size_t hostBytes = N * sizeof(float);
	check(hipMalloc(&device, hostBytes));
	check(hipMemcpy(device, host, hostBytes, hipMemcpyHostToDevice));
	return device;
}

__host__ float* pinHostAndSendDevice(const size_t N, float* host) {
	float* device;
	const size_t hostBytes = N * sizeof(float);
	check(hipHostRegister(host, hostBytes, hipHostRegisterDefault));
	check(hipMalloc(&device, hostBytes));
	check(hipMemcpy(device, host, hostBytes, hipMemcpyHostToDevice));
	return device;
}

__host__ void recvDeviceUnpinHost(float* device, float* host, const size_t N) {
	check(hipMemcpy(host, device, N * sizeof(float), hipMemcpyDeviceToHost));
	hipFree(device);
	hipHostUnregister(host);
}

__host__ void unpinHost(float* device, float* host) {
	hipFree(device);
	hipHostUnregister(host);
}

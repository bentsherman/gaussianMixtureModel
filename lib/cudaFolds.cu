#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"
#include "cudaFolds.hu"

// ----------------------------------------------------------------------------
// Find sum of a vector array
// ----------------------------------------------------------------------------

__device__ void devVecAdd(size_t pointDim, float* dest, float* src) {
	for(size_t i = 0; i < pointDim; ++i) {
		dest[i] += src[i];
	}
}

__global__ void kernElementWiseSum(const size_t numPoints, const size_t pointDim, float* dest, float* src) {
	// Called to standardize arrays to be a power of two

	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	if(i < numPoints) {
		devVecAdd(pointDim, &dest[i * pointDim], &src[i * pointDim]);
	}
}

__global__ void kernBlockWiseSum(const size_t numPoints, const size_t pointDim, float* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	// call repeatedly for each dimension where dest is assumed to begin at dimension d

	__shared__ float blockSum[1024];

	if(threadIdx.x >= numPoints) {
		blockSum[threadIdx.x] = 0;
	} else {
		blockSum[threadIdx.x] = dest[i * pointDim];
	}

	__syncthreads();

	// Do all the calculations in block shared memory instead of global memory.
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
		blockSum[threadIdx.x] += blockSum[threadIdx.x + s];
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		// Just do one global write
		dest[i * pointDim] = blockSum[0];
	}	
}

__global__ void kernMoveMem(const size_t numPoints, const size_t pointDim, const size_t s, float* A) {
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	// Before
	// [abc......] [def......] [ghi......] [jkl......]

	// shared memory
	// [adgj.....]

	// After
	// [a..d..g..] [j........] [ghi......] [.........]

	__shared__ float mem[1024];
	mem[threadIdx.x] = A[s * i * pointDim];
	__syncthreads();
	A[i * pointDim] = mem[threadIdx.x];
}

__host__ void cudaArraySum(hipDeviceProp_t* deviceProp, size_t numPoints, const size_t pointDim, float* device_A, hipStream_t stream) {
	assert(deviceProp != NULL);
	assert(numPoints > 0);
	assert(pointDim > 0);
	assert(device_A != NULL);

	size_t M = largestPowTwoLessThanEq(numPoints);
	if(M != numPoints) {
		dim3 block , grid;
		calcDim(M, deviceProp, &block, &grid);
		kernElementWiseSum<<<grid, block, 0, stream>>>(
			numPoints - M, pointDim, device_A, device_A + M * pointDim
		);
		numPoints = M;
	}

	while(numPoints > 1) {
		dim3 block, grid;
		calcDim(numPoints, deviceProp, &block, &grid);

		for(size_t d = 0; d < pointDim; ++d) {
			kernBlockWiseSum<<<grid, block, 0, stream>>>(numPoints, pointDim, device_A + d);
			
			if(numPoints > block.x) {
				dim3 block2, grid2;
				calcDim(grid.x, deviceProp, &block2, &grid2);
				kernMoveMem<<<grid2, block2, 0, stream>>>(numPoints, pointDim, block.x, device_A + d);
			}
		}

		numPoints /= block.x;
	}
}

// ----------------------------------------------------------------------------
// Find maximum of a scalar array
// ----------------------------------------------------------------------------

__global__ void kernElementWiseMax(const size_t numPoints, float* dest, float* src) {
	// Called to standardize arrays to be a power of two

	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	if(i < numPoints) {
		if(dest[i] < src[i]) {
			dest[i] = src[i];
		}
	}
}

__global__ void kernBlockWiseMax(const size_t numPoints, float* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	__shared__ float blockMax[1024];

	if(threadIdx.x >= numPoints) {
		blockMax[threadIdx.x] = -INFINITY;
	} else {
		blockMax[threadIdx.x] = dest[i];
	}

	__syncthreads();

	// Do all the calculations in block shared memory instead of global memory.
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
		if(blockMax[threadIdx.x] < blockMax[threadIdx.x + s]) {
			blockMax[threadIdx.x] = blockMax[threadIdx.x + s];
		}
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		// Just do one global write
		dest[i] = blockMax[0];
	}
}

__host__ void cudaArrayMax(hipDeviceProp_t* deviceProp, size_t numPoints, float* device_A, hipStream_t stream) {
	assert(deviceProp != NULL);
	assert(numPoints > 0);
	assert(device_A != NULL);

	size_t M = largestPowTwoLessThanEq(numPoints);
	if(M != numPoints) {
		dim3 block , grid;
		calcDim(M, deviceProp, &block, &grid);
		kernElementWiseMax<<<grid, block, 0, stream>>>(
			numPoints - M, device_A, device_A + M
		);
		numPoints = M;
	}

	while(numPoints > 1) {
		dim3 block, grid;
		calcDim(numPoints, deviceProp, &block, &grid);

		kernBlockWiseMax<<<grid, block, 0, stream>>>(numPoints, device_A);
		
		if(numPoints > block.x) {
			dim3 block2, grid2;
			calcDim(grid.x, deviceProp, &block2, &grid2);
			kernMoveMem<<<grid2, block2, 0, stream>>>(numPoints, 1, block.x, device_A);
		}

		numPoints /= block.x;
	}
}

